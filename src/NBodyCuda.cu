#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "main.h"
#include "NBody.cuh"
#include <algorithm>
#include <math.h>
#include <iostream>

#define BLOCKSIZE (128)

using namespace std;


/*cuda shared memory*/
template<class T>
struct SharedMemory
{
	__device__ inline operator       T *()
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}

	__device__ inline operator const T *() const
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}
};

double getRandom(double min, double max)
{
	float r = (float)rand() / RAND_MAX;
	return r * (max - min) + min;
}

struct body *initializeNBodyCuda(char method)
{
	/**
	initialize the bodies, then copy to the CUDA device memory
	return the device pointer so that it can be reused in the NBodyTimestepCuda function.
	*/

	/*Using CUDA Unified Memory to automatically handle memory operation*/
	body *bodies;
	const double PI = 3.14159265;
	hipMallocManaged((void **) &bodies, sizeof(body) * (NUM_BODIES+1));/*using bodies[NUM_BODIES] to store cursor body*/
	if(method == '0')
	{
		cout<<"Initialized by random position and mass"<<endl;
		for (auto i = 0; i < NUM_BODIES+1; i++)
		{
			bodies[i].m = getRandom(0,1.0);
			bodies[i].x = getRandom(-1,1);
			bodies[i].y = getRandom(-1,1);
			bodies[i].vx = 0;
			bodies[i].vy = 0;
		}
	}
	else
	{
		cout<<"Initialized by random velocity and mass"<<endl;
		for (auto i = 0; i < NUM_BODIES+1; i++)
		{
			bodies[i].m = getRandom(0,1.0);
			bodies[i].x = 0;
			bodies[i].y = 0;
			auto radius = getRandom(0,0.05);
			auto theta = getRandom(0,2*PI);
			bodies[i].vx = radius*cos(theta);
			bodies[i].vy = radius*sin(theta);
		}
	}
	return bodies;
}

unsigned char* initCanvas()
{
	/*Using CUDA Unified Memory to automatically handle memory operation, such that we can parallel rasterasize function*/
	unsigned char* buffer;
	hipMallocManaged((void **) &buffer, sizeof(unsigned char) * SCREEN_WIDTH * SCREEN_HEIGHT * 3);
	return buffer;
}

__global__
void rasterizeKernel(unsigned char* buffer, body* bodies)
{
	/*rasterize kernel*/
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= NUM_BODIES)
	{
		return;
	}
	int x = (int) lround(SCREEN_WIDTH * ((bodies[index].x + 1) / 2.0));
	int y = (int) lround(SCREEN_HEIGHT * ((bodies[index].y + 1) / 2.0));
	buffer[x * SCREEN_WIDTH * 3 + y * 3 + 0] = 0XFF;
	buffer[x * SCREEN_WIDTH * 3 + y * 3 + 1] = 0XFE;
	buffer[x * SCREEN_WIDTH * 3 + y * 3 + 2] = 0XE5;
}

__global__
void trailKernel(unsigned char* buffer)
{
	/*trail effection kernel*/
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= SCREEN_WIDTH * SCREEN_HEIGHT * 3)
	{
		return;
	}
	buffer[index] = (unsigned char)(buffer[index] * 0.8);
}


void rasterize(struct body *bodies, unsigned char *buffer)
{
	/**
	rasterize the bodies from x,y: (-1,-1) to (1,1) according to some kind of formula

	Note: You can change the code for better visualization
	As the following code can be parallelized, you can optimize this routine with CUDA.

	\param bodies A collection of bodies (located on the device).
	\param buffer the RGB buffer for screen display (located on the host).
	*/
	/*trail effection*/
	dim3 blockSize(BLOCKSIZE);
	dim3 gridSize((SCREEN_WIDTH * SCREEN_HEIGHT * 3 + blockSize.x - 1) / blockSize.x);
	trailKernel<<< gridSize, blockSize>>>(buffer);
	hipDeviceSynchronize();
	/*rasterize*/
	dim3 gridSize2((NUM_BODIES + blockSize.x - 1) / blockSize.x);
	rasterizeKernel<<< gridSize2, blockSize>>>(buffer,bodies);
	hipDeviceSynchronize();
}


void freeCudaMem(void* p)
{
	/*ferr malloced memory mannually*/
	hipFree(p);
}

__device__
float2 bodyBodyInteraction(body selfBody, body body2, float2 acc)
{
	/*calculate accelaration between 2 bodies*/
	float2 r;
	r.x = body2.x - selfBody.x;
	r.y = body2.y - selfBody.y;
	float disSquare = r.x * r.x + r.y * r.y + eps * eps;
	float tmp = G * body2.m / rsqrt(disSquare * disSquare * disSquare);
	acc.x += r.x * tmp;
	acc.y += r.y * tmp;
	return acc;
}


__device__
float2 computeBodyAccel(body selfBody, body* bodies, int numTiles)
{
	/*compute the body's accelaration under all the other bodies' effection*/
	body *sharedBodies = SharedMemory<body>();
	float2 acc; acc.x =0; acc.y=0;
	for (int tile = 0; tile < numTiles; tile++)
	{
		sharedBodies[threadIdx.x] = bodies[tile * blockDim.x + threadIdx.x];
		__syncthreads();
		#pragma unroll 128
		for (unsigned int counter = 0; counter < blockDim.x; counter++)
		{
			acc = bodyBodyInteraction(selfBody, sharedBodies[counter], acc);
		}
		__syncthreads();
	}
	return acc;
}


__global__ void
integrateBodies(body* bodies, float deltaTime,int numTiles)
{
	/*N-Body kernel*/
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= NUM_BODIES)
	{
		return;
	}
	bodies[index].x += bodies[index].vx *deltaTime;
	bodies[index].y += bodies[index].vy *deltaTime;

	body b1 = bodies[index];
	float2 accel = computeBodyAccel(b1,bodies,numTiles);
	bodies[index].vx += accel.x * deltaTime;
	bodies[index].vy += accel.y * deltaTime;
	// solve boundary problem
	if (bodies[index].x > 1 || bodies[index].x < -1)
	{
		bodies[index].vx *= (-collision_damping);
		bodies[index].x = (bodies[index].x > 0 ? 1.0 : -1.0);
	}
	if (bodies[index].y > 1 || bodies[index].y < -1)
	{
		bodies[index].vy *= (-collision_damping);
		bodies[index].y = (bodies[index].y > 0 ? 1.0 : -1.0);
	}
}



void NBodyTimestepCuda(struct body *bodies, float rx, float ry, bool cursor)
{
	/**
	Compute a time step on the CUDA device.
	TODO: correctly manage the device memory, compute the time step with proper block/threads

	\param bodies A collection of bodies (located on the device).
	\param rx position x of the cursor.
	\param ry position y of the cursor.
	\param cursor Enable the mouse interaction if true (adding a weight = cursor_weight body in the computation).
	*/
	float timeStep = 1;
	/*cursor body*/
	bodies[NUM_BODIES].m = cursor_weight * (float) cursor; // convert bool condition to scale multiply to avoid branch selection
	bodies[NUM_BODIES].x = rx;
	bodies[NUM_BODIES].y = ry;
	bodies[NUM_BODIES].vx = 0;
	bodies[NUM_BODIES].vy = 0;
	/*N-Body cuda kernel exec*/
	int numTiles = (NUM_BODIES + BLOCKSIZE) / BLOCKSIZE;/*add 1 to store cursor body*/
	dim3 blockSize(BLOCKSIZE);
	dim3 gridSize(numTiles);
	int sharedMemSize = BLOCKSIZE * sizeof(body);
	integrateBodies<<< gridSize, blockSize, sharedMemSize >>>(bodies,timeStep,numTiles);
	hipDeviceSynchronize();
}
